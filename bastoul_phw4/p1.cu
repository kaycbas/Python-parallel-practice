#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hipblas.h>
#include <time.h>

#define n 1024
#define BLOCK_SIZE 8
#define GRID_SIZE 1024/BLOCK_SIZE

typedef struct {
	int width;
	int height;
	int* elements;
} Matrix;

__global__ void matrix_mult(Matrix a, Matrix b, Matrix c) {

	int id_x = blockIdx.x * blockDim.x + threadIdx.x;
	int id_y = blockIdx.y * blockDim.y + threadIdx.y;
	
	//if(row > a.height || col > b.width) return;

	int local_c = 0;
	for (int i=0; i<n; i++) {
		local_c += (a.elements[id_x*n+i]) * (b.elements[i*n+id_y]);
	}
	c.elements[id_x*n+id_y] = local_c;
}

int main () {
	int i;

	Matrix a;
	Matrix b;
	Matrix c;

	a.height = n;
	a.width = n;
	b.height = n;
	b.width = n;
	c.height = n;
	c.width = n;

	a.elements = (int*) malloc (a.width * a.height * sizeof(int));
	b.elements = (int*) malloc (b.width * b.height * sizeof(int));
	c.elements = (int*) malloc (c.width * c.height * sizeof(int));

	for (i=0; i<n*n; i++) {
		a.elements[i] = 1;
		b.elements[i] = 2;
		c.elements[i] = 0;
	}


	Matrix gpu_a;
	Matrix gpu_b;
	Matrix gpu_c;

	hipError_t err = hipMalloc(&gpu_a.elements, sizeof(int)*n*n);
	printf("CUDA malloc A: %s\n",hipGetErrorString(err));
	err = hipMalloc(&gpu_b.elements, sizeof(int)*n*n);
	printf("CUDA malloc B: %s\n",hipGetErrorString(err));
	err = hipMalloc(&gpu_c.elements, sizeof(int)*n*n);
	printf("CUDA malloc C: %s\n",hipGetErrorString(err));


	err = hipMemcpy(gpu_a.elements, a.elements, sizeof(int)*n*n, hipMemcpyHostToDevice);
	printf("Copy A onto device: %s\n",hipGetErrorString(err));
	err = hipMemcpy(gpu_b.elements, b.elements, sizeof(int)*n*n, hipMemcpyHostToDevice);
	printf("Copy B onto device: %s\n",hipGetErrorString(err));
	err = hipMemcpy(gpu_c.elements, c.elements, sizeof(int)*n*n, hipMemcpyHostToDevice);
	printf("Copy C onto device: %s\n",hipGetErrorString(err));


	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(GRID_SIZE, GRID_SIZE);

	struct timespec start, stop; 
    double time;

    if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
	matrix_mult<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
	hipDeviceSynchronize();

	err = hipMemcpy(c.elements, gpu_c.elements, sizeof(int)*n*n, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));

	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("time is %f ns\n", time*1e9);

	//for (i=0; i<n*n; ++i) {
	std::cout << "c[451][451] = " << c.elements[451*n+451] << std::endl;
	//}

	free(a.elements);
	free(b.elements);
	free(c.elements);
	hipFree(gpu_a.elements);
	hipFree(gpu_b.elements);
	hipFree(gpu_c.elements);

	return 0;
}

